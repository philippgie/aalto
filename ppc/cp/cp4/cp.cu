#include <chrono>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
using namespace std::chrono;

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

float average(const float *data, const int length);
float std_dev(const float *data, const int length);
float squared_sum(const float *data, const int length);


/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
__global__ void mykernel(int nx, int ny, const float *data, float *result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    //if (i >= ny || j >= n)
    //    return;


    float res=0.0;
    for (int k = 0; k < nx; k++) {
      float x = data[k+ i* nx];
      float y = data[k+ j* nx];
      res += x * y;
    }
    result[ny * i + j] = res;
}

void correlate(int ny, int nx, const float *data, float *result) {
    // Allocate memory & copy data to GPU
    float* dataGPU = NULL;
    CHECK(hipMalloc((void**)&dataGPU, nx * ny * sizeof(float)));
    float* resultGPU = NULL;
    CHECK(hipMalloc((void**)&resultGPU, nx * ny * sizeof(float)));

    float *data_norm = new float[nx * ny]{0.0};
    for (int y = 0; y < ny; y++) {
        float *values = &data_norm[y * nx];

        float mean = average(&data[y * nx], nx);
        // std::cout << "Mean of simple norm: " << mean << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] = data[y * nx + i] - mean;
        }

        float s = std_dev(values, nx);
        // std::cout << "Standard deviation of simple norm: " << s << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] /= s;
        }

        float factor = std::sqrt(squared_sum(values, nx));
        // std::cout << "Factor of squared norm: " << factor << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] /= factor;
        }
    }

    CHECK(hipMemcpy(dataGPU, data_norm, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(nx, dimBlock.x), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(nx, ny, dataGPU, resultGPU);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resultGPU, nx * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resultGPU));

}

inline float average(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += static_cast<float>(data[x]);
  }
  return aggr / (float)length;
}

inline float std_dev(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += data[x] * data[x];
  }
  return aggr / (float)length;
}

inline float squared_sum(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += data[x] * data[x];
  }
  return aggr;
}

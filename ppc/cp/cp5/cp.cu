#include <chrono>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
using namespace std::chrono;


static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

float average(const float *data, const int length);
float std_dev(const float *data, const int length);
float squared_sum(const float *data, const int length);


/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
__global__ void mykernel(int nx, int ny, const float *data, const float *data_t, float *result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny )
        return;
    if (i<j)
      result[ny * j + i] = 0.0;


    float res=0.0;
    for (int k = 0; k < nx; k++) {
      float x = data[k+ j* nx];
      float y = data_t[i+ k* ny];
      res += x * y;
    }
    result[ny * j + i] = res;
    //printf("%d is %f\n", ny*j+i,res);
}

void correlate(int ny, int nx, const float *data, float *result) {
    // Allocate memory & copy data to GPU
    float* dataGPU = NULL;
    CHECK(hipMalloc((void**)&dataGPU, nx * ny * sizeof(float)));
    float* dataTGPU = NULL;
    CHECK(hipMalloc((void**)&dataTGPU, nx * ny * sizeof(float)));
    float* resultGPU = NULL;
    CHECK(hipMalloc((void**)&resultGPU, ny * ny * sizeof(float)));

    float *data_norm = new float[nx * ny]{0.0};
    auto start = high_resolution_clock::now();

    for (int y = 0; y < ny; y++) {
        float *values = &data_norm[y * nx];

        float mean = average(&data[y * nx], nx);
        // std::cout << "Mean of simple norm: " << mean << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] = data[y * nx + i] - mean;
        }

        float s = std_dev(values, nx);
        // std::cout << "Standard deviation of simple norm: " << s << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] /= s;
        }

        float factor = std::sqrt(squared_sum(values, nx));
        // std::cout << "Factor of squared norm: " << factor << std::endl;
        for (int i = 0; i < nx; i++) {
          values[i] /= factor;
        }
    }
        auto stop = high_resolution_clock::now();

    auto duration = duration_cast<milliseconds>(stop - start);
 
    std::cout << "Time taken by function: "
         << duration.count() << " milliseconds" << std::endl;

    float *data_t = new float[nx * ny]{0.0};
    for(int i = 0; i<nx;i++){
      for(int j = 0; j<ny;j++){
        data_t[j+ny*i]=data_norm[i+nx*j];
      }
    }
    //for(int i = 0; i<nx*ny;i++)
    //  std::cout<<data_t[i]<<std::endl;

    CHECK(hipMemcpy(dataGPU, data_norm, nx * ny * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dataTGPU, data_t, nx * ny * sizeof(float), hipMemcpyHostToDevice));
    delete []data_norm;
    delete []data_t;


    // Run kernel
    dim3 dimBlock(16, 16);
    //dim3 dimGrid(divup(nx, dimBlock.x), divup(ny, dimBlock.y));
    dim3 dimGrid(divup(ny, dimBlock.y), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(nx, ny, dataGPU, dataTGPU, resultGPU);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(dataTGPU));
    CHECK(hipFree(resultGPU));


}

inline float average(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += static_cast<float>(data[x]);
  }
  return aggr / (float)length;
}

inline float std_dev(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += data[x] * data[x];
  }
  return aggr / (float)length;
}

inline float squared_sum(const float *data, const int length) {
  float aggr = 0.0;
  for (int x = 0; x < length; x++) {
    aggr += data[x] * data[x];
  }
  return aggr;
}
